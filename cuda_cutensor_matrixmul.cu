/* 
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions are met:
 *   - Redistributions of source code must retain the above copyright notice,
 *     this list of conditions and the following disclaimer.
 *   - Redistributions in binary form must reproduce the above copyright notice,
 *     this list of conditions and the following disclaimer in the documentation
 *     and/or other materials provided with the distribution.
 *   - Neither the names of copyright holders nor the names of its contributors
 *     may be used to endorse or promote products derived from this software 
 *     without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS "AS IS"
 * AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE
 * LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
 * CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
 * SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS 
 * INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN
 * CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE)
 * ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE
 * POSSIBILITY OF SUCH DAMAGE.
 */

/**
 * ---------------------------------------------------------------------------
 * Matrix Multiplication via cuTENSOR: GPU Tensor Contraction Example
 * ---------------------------------------------------------------------------
 *
 * This program demonstrates the application of high-performance tensor
 * contraction for matrix multiplication on NVIDIA GPUs via the cuTENSOR library.
 * The implementation initializes random input matrices, performs the contraction
 * corresponding to C = alpha * A * B + beta * C using cuTENSOR's optimized APIs,
 * and outputs the results.
 * 
 * The code provides a reproducible computational workflow suitable for research,
 * benchmarking, or advanced educational purposes. 
 */


#include <cstdio>
#include <cstdlib>
#include <cassert>
#include <fstream>
#include <hip/hip_runtime.h>
#include <hiptensor.h>
#include <unordered_map>
#include <vector>

// Error handling macros
#define HANDLE_ERROR(x) { \
  hiptensorStatus_t err = x; \
  if (err != HIPTENSOR_STATUS_SUCCESS) { \
    printf("cuTENSOR error: %s\n", hiptensorGetErrorString(err)); exit(-1); \
  } \
}

#define HANDLE_CUDA_ERROR(x) { \
    hipError_t err = x; \
    if (err != hipSuccess) { \
        printf("CUDA error: %s\n", hipGetErrorString(err)); exit(-1); \
    } \
}

int main()
{
    typedef float floatType;
    // Matrix sizes: C[m, k] = sum_h A[m, h] * B[h, k]
    const int M = 96, H = 64, K = 128;

    // Mode labels ('m', 'h', 'k')
    std::vector<int> modeA{'m','h'};
    std::vector<int> modeB{'h','k'};
    std::vector<int> modeC{'m','k'};
    int nmodeA = modeA.size();
    int nmodeB = modeB.size();
    int nmodeC = modeC.size();

    // Extents mapping ('m','h','k')
    std::unordered_map<int, int64_t> extent;
    extent['m'] = M;
    extent['h'] = H;
    extent['k'] = K;

    // Create vectors of extents for each tensor
    std::vector<int64_t> extentA, extentB, extentC;
    for(auto mode : modeA) extentA.push_back(extent[mode]);
    for(auto mode : modeB) extentB.push_back(extent[mode]);
    for(auto mode : modeC) extentC.push_back(extent[mode]);

    // Number of elements and allocation
    size_t elementsA = 1, elementsB = 1, elementsC = 1;
    for(auto mode : modeA) elementsA *= extent[mode];
    for(auto mode : modeB) elementsB *= extent[mode];
    for(auto mode : modeC) elementsC *= extent[mode];
    size_t sizeA = sizeof(floatType) * elementsA;
    size_t sizeB = sizeof(floatType) * elementsB;
    size_t sizeC = sizeof(floatType) * elementsC;

    // Allocate on host
    floatType *A = (floatType*) malloc(sizeA);
    floatType *B = (floatType*) malloc(sizeB);
    floatType *C = (floatType*) malloc(sizeC);
    for(int64_t i = 0; i < elementsA; i++)
        A[i] = (((float) rand())/RAND_MAX - 0.5f)*10;
    for(int64_t i = 0; i < elementsB; i++)
        B[i] = (((float) rand())/RAND_MAX - 0.5f)*10;
    for(int64_t i = 0; i < elementsC; i++)
        C[i] = 0.0f;

    // Allocate on device
    void *A_d, *B_d, *C_d;
    HANDLE_CUDA_ERROR(hipMalloc(&A_d, sizeA));
    HANDLE_CUDA_ERROR(hipMalloc(&B_d, sizeB));
    HANDLE_CUDA_ERROR(hipMalloc(&C_d, sizeC));

    // Copy to device
    HANDLE_CUDA_ERROR(hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice));
    HANDLE_CUDA_ERROR(hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice));

    // Alignment for cuTENSOR (128 bytes)
    const uint32_t kAlignment = 128;
    assert(uintptr_t(A_d) % kAlignment == 0);
    assert(uintptr_t(B_d) % kAlignment == 0);
    assert(uintptr_t(C_d) % kAlignment == 0);

    // Initialize cuTENSOR handle
    hiptensorHandle_t handle;
    HANDLE_ERROR(hiptensorCreate(&handle));

    // Create Tensor Descriptors
    hiptensorTensorDescriptor_t descA, descB, descC;
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descA, nmodeA, extentA.data(), NULL, HIPTENSOR_R_32F, kAlignment));
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descB, nmodeB, extentB.data(), NULL, HIPTENSOR_R_32F, kAlignment));
    HANDLE_ERROR(hiptensorCreateTensorDescriptor(handle, &descC, nmodeC, extentC.data(), NULL, HIPTENSOR_R_32F, kAlignment));

    // Create Contraction Descriptor
    hiptensorOperationDescriptor_t contractionDesc;
    hiptensorComputeDescriptor_t descCompute = HIPTENSOR_COMPUTE_DESC_32F;
    HANDLE_ERROR(hiptensorCreateContraction(handle,
        &contractionDesc,
        descA, modeA.data(), HIPTENSOR_OP_IDENTITY,
        descB, modeB.data(), HIPTENSOR_OP_IDENTITY,
        descC, modeC.data(), HIPTENSOR_OP_IDENTITY,
        descC, modeC.data(),
        descCompute));

    // Algorithm & Plan Preferences
    const hiptensorAlgo_t algo = HIPTENSOR_ALGO_DEFAULT;
    hiptensorPlanPreference_t planPref;
    HANDLE_ERROR(hiptensorCreatePlanPreference(handle, &planPref, algo, HIPTENSOR_JIT_MODE_NONE));

    // Workspace Estimation
    uint64_t workspaceSizeEstimate = 0, actualWorkspaceSize = 0;
    HANDLE_ERROR(hiptensorEstimateWorkspaceSize(handle, contractionDesc, planPref, HIPTENSOR_WORKSPACE_DEFAULT, &workspaceSizeEstimate));

    // Create Plan
    hiptensorPlan_t plan;
    HANDLE_ERROR(hiptensorCreatePlan(handle, &plan, contractionDesc, planPref, workspaceSizeEstimate));

    // Query actual workspace
    HANDLE_ERROR(hiptensorPlanGetAttribute(handle, plan, HIPTENSOR_PLAN_REQUIRED_WORKSPACE, &actualWorkspaceSize, sizeof(actualWorkspaceSize)));

    void *workspace = nullptr;
    if (actualWorkspaceSize > 0) {
        HANDLE_CUDA_ERROR(hipMalloc(&workspace, actualWorkspaceSize));
        assert(uintptr_t(workspace) % 128 == 0);
    }

    // CUDA stream
    hipStream_t stream;
    HANDLE_CUDA_ERROR(hipStreamCreate(&stream));

    // Do tensor contraction: C = alpha * A * B + beta * C
    float alpha = 1.0f, beta = 0.0f;
    HANDLE_ERROR(hiptensorContract(handle, plan,
        (void*) &alpha, A_d, B_d,
        (void*) &beta, C_d, C_d,
        workspace, actualWorkspaceSize, stream));

    // Copy result to host
    HANDLE_CUDA_ERROR(hipMemcpy(C, C_d, sizeC, hipMemcpyDeviceToHost));

    // Save results to a text file
    std::ofstream fout("results/matrix_output.txt");
    if(fout.is_open()) {
        for(int m=0; m<M; ++m){
            for(int k=0; k<K; ++k) {
                fout << C[m*K + k] << " ";
            }
            fout << "\n";
        }
        fout.close();
        printf("Results written to results/matrix_output.txt\n");
    }
    else {
        printf("Unable to open results/matrix_output.txt for writing!\n");
    }

    // Cleanup
    hiptensorDestroyPlan(plan);
    hiptensorDestroyOperationDescriptor(contractionDesc);
    hiptensorDestroyTensorDescriptor(descA);
    hiptensorDestroyTensorDescriptor(descB);
    hiptensorDestroyTensorDescriptor(descC);
    hiptensorDestroy(handle);
    hipStreamDestroy(stream);
    if(workspace) hipFree(workspace);
    hipFree(A_d); hipFree(B_d); hipFree(C_d);
    free(A); free(B); free(C);
    return 0;
}

